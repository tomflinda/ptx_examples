
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include "hipblas.h"

__global__ void helloFromGPU (void) {
  __shared__ half aTile[2*8*8];

  int tidx = threadIdx.x + blockDim.x * threadIdx.y;
  if (tidx == 0) {
    for (int i = 0; i < 2*8*8; ++i) {
        aTile[i] = i;
    }
  }
  __syncthreads();

  int aTile_index = tidx * 8;
  uint32_t my_register[2];
  uint32_t smem = __cvta_generic_to_shared(aTile+aTile_index);
  asm("ldmatrix.sync.aligned.m8n8.x2.trans.shared.b16 { %0, %1 }, [ %2 ];\n"
  : "=r"(my_register[0]), "=r"(my_register[1])
  : "r"(smem)
  );

  if (tidx == 4) {
    for (int i = 0; i < 2; i++) {
        half * tmp = (half*)(&(my_register[i]));
        printf("%f\n", (float)(tmp[0]));
        printf("%f\n", (float)(tmp[1]));
    }
  }
}

int main(void) {
uint3 block = {32,1,1};
uint3 grid = {1,1,1};
helloFromGPU <<<grid, block>>>();

hipDeviceReset();
return 0;
}